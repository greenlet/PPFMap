#include "hip/hip_runtime.h"
// #include <PPFMap/common.h>
#include <PPFMap/ppf_cuda_calls.h>

ppfmap::Map::Ptr 
ppfmap::cuda::setPPFMap(const float3 *points, 
                        const float3 *normals,
                        const size_t n,
                        const float disc_dist,
                        const float disc_angle) {

    thrust::host_vector<float3> h_points(points, points + n);
    thrust::host_vector<float3> h_normals(normals, normals + n);

    return boost::shared_ptr<Map>(new Map(h_points, h_normals, disc_dist, disc_angle));
}
